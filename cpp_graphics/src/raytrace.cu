#include "hip/hip_runtime.h"
#include "raytrace.cuh"
#include <iostream>

#include <stdio.h>

inline __host__ __device__ float3 vec_to_float3(Vector3 v)
{
    return make_float3(v.x, v.y, v.z);
}

inline __host__ __device__ float3 operator-(float3 &a)
{
    return make_float3(-a.x, -a.y, -a.z);
}

inline __host__ __device__ float3 operator+(const float3 &a, const float3 &b) {
  return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

inline __host__ __device__ float3 operator-(const float3 &a, const float3 &b) {
  return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

inline __host__ __device__ float3 operator*(const float3 &a, const float3 &b) {
  return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}

inline __host__ __device__ float3 operator*(const float3 &a, const float &s) {
  return make_float3(a.x*s, a.y*s, a.z*s);
}

inline __host__ __device__ float3 operator*(const float &s, const float3 &a) {
  return make_float3(a.x*s, a.y*s, a.z*s);
}

inline __host__ __device__ void operator+=(float3 &a, float3 b)
{
    a.x += b.x;
    a.y += b.y;
    a.z += b.z;
}

inline __host__ __device__ float dot(float3 a, float3 b)
{
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

inline __host__ __device__ float3 cross(float3 a, float3 b)
{
    return make_float3(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x);
}

inline __host__ __device__ float3 normalize(float3 v)
{
    float invLen = rsqrtf(dot(v, v));
    return v * invLen;
}

inline __host__ __device__ float magnitude(float3 v)
{
    return sqrt(dot(v, v));
}

inline __host__ __device__ float3 reflect(float3 dir, float3 normal){
	return (dot(normalize(dir),  normalize(normal)) * 2.0 * normalize(normal)) - normalize(dir);
}

namespace Raytrace {
	struct Triangle {
		float3 v0, v1, v2, n, color;
		float diffuse, specular, ambient, shinyness, reflectivity, transparency;
		Triangle() {}
		Triangle(float3 v0, float3 v1, float3 v2, float3 color, float diffuse, float specular, float ambient, float shinyness, float reflectivity, float transparency) : v0(v0), v1(v1), v2(v2), n(normalize(cross(v1-v0, v2-v0))), color(color), diffuse(diffuse), specular(specular), ambient(ambient), shinyness(shinyness), reflectivity(reflectivity), transparency(transparency) {}

		__device__ float intersect(float3 dir, float3 origin, float3 &hitPoint, float3 &hitNormal){
			//Cases where ray does not intersect: normal dot dir is close to 0 (parallel), t  < 0 (triangle behind ray), N dot (edge X v(0..2)) (passes a side)

			float nDotDir = dot(n, dir);
			
			if(abs(nDotDir) < 2e-8){
				return -1;
			}
			
			float t = -(dot(n, origin) - dot(n, v0)) / dot(n, dir);
			
			if(t < 0){
				return -1;
			}

			hitPoint = origin + dir * t; 
			hitNormal = n;
			float depth = magnitude(hitPoint-origin);

			if(dot(n, cross(v1-v0, hitPoint-v0)) < 0 || dot(n, cross(v2-v1, hitPoint-v1)) < 0 || dot(n, cross(v0-v2, hitPoint- v2)) < 0){
				return -1;
			}

			return depth;
		}
	};
	

	struct Light{
		float3 point, color;
		float intensity;
		Light(){}
		Light(float3 point, float3 color, float intensity) : point(point), color(color), intensity(intensity) {}
	};

	void render(int width, int height, float fx, float fy, Vector3 origin, Vector3 rotation, std::vector<CGLTri*> objects, std::vector<CGLLight*> lights, int** pixels){
		int h_numTris = objects.size();
		Triangle h_tris[h_numTris];
		for(int i = 0; i < objects.size(); i++){
			Triangle tri = Triangle(vec_to_float3(objects[i]->p0), vec_to_float3(objects[i]->p1), vec_to_float3(objects[i]->p2), vec_to_float3(objects[i]->color), objects[i]->diffuse, objects[i]->specular, objects[i]->ambient, objects[i]->shinyness, objects[i]->reflectivity, objects[i]->transparency);
			h_tris[i] = tri;
		}

		int h_numLights = lights.size();
		Light h_lights[h_numLights];
		for(int i = 0; i < lights.size(); i++){
			Light light = Light(vec_to_float3(lights[i]->point), vec_to_float3(lights[i]->color), lights[i]->intensity);
			h_lights[i] = light;
		}

		Triangle *d_tris;
		Light *d_lights;
		int *d_pixels;
		int *h_pixels = (int*)malloc(width*height*sizeof(int));
		
		hipMalloc(&d_tris, sizeof(h_tris));
		hipMalloc(&d_lights, sizeof(h_lights));
		hipMalloc(&d_pixels, width*height*sizeof(int));
		
		hipMemcpy(d_tris, h_tris, sizeof(h_tris), hipMemcpyHostToDevice);
		hipMemcpy(d_lights, h_lights, sizeof(h_lights), hipMemcpyHostToDevice);

		// render each pixel
		int N = width*height;
    	int thr_per_blk = 256;
    	int blk_in_grid = ceil( float(N) / thr_per_blk );

		float3 d_origin = make_float3(origin.x, origin.y, origin.z);
		float3 d_rotation = make_float3(rotation.x, rotation.y, rotation.z);

		// std::cout << thr_per_blk << std::endl;

		trace<<<blk_in_grid, thr_per_blk>>>(fx, fy, width, height, d_origin, d_rotation, d_tris, h_numTris, d_lights, h_numLights, d_pixels);

		hipMemcpy(h_pixels, d_pixels, width*height*sizeof(int), hipMemcpyDeviceToHost);

		for(int id = 0; id < width*height; id++){
			int px = id % width;
			int py = id / width;

			pixels[px][py] = h_pixels[id];

			// std::cout << px << " " << py << " " <<  h_pixels[id] << std::endl;
		}
		
		hipFree(d_tris);
	    hipFree(d_lights);
	}

	__device__ unsigned long createRGB(int r, int g, int b)
	{   
		return ((r & 0xff) << 16) + ((g & 0xff) << 8) + (b & 0xff);
	}

	__global__ void trace(float fx, float fy, int width, int height, float3 origin, float3 rotation, Triangle tris[], int numTris, Light lights[], int numLights, int* pixels)
	{
		int id = blockDim.x * blockIdx.x + threadIdx.x;
		int px = id % width;
		int py = id / width;

		float x =  px - width/2.0;
		float y =  height/2.0 - py;
		float dx = x/-fx;
		float dy = y/-fy;
		float3 dir = normalize(make_float3(dx * cos(rotation.x) + 1 * sin(rotation.x), dy, -dx * sin(rotation.x) + 1 * cos(rotation.x)));
		

		int color = 0;
		raytrace(dir, origin, tris, numTris, lights, numLights, color, 0);
		
		pixels[id] = color;
	}

	__device__ void raytrace(float3 dir, float3 origin, Triangle tris[], int numTris, Light lights[], int numLights, int &color, int step){
		float3 hitPoint;
		float3 N;
		int hitIndex;
		float depth = 0;

		color = createRGB(0, 0, 0);

		cast(dir, origin, tris, numTris, hitPoint, N, hitIndex, depth);
		if(depth > 0 && depth > 0.01){
			if(step > 2){
				color = createRGB(min(int(tris[hitIndex].color.x), 255), min(int(tris[hitIndex].color.y), 255), min(int(tris[hitIndex].color.z), 255));
			}
			else{
				float3 V = normalize(origin-hitPoint);

				float3 diffuse = make_float3(0, 0, 0);
				float3 specular = make_float3(0, 0, 0);

				float3 hitOffset = (dot(dir, N) < 0) ? (hitPoint + N * 0.001) : (hitPoint - N * 0.001);
				for(int i = 0; i < numLights; i++){
					float3 L = normalize(lights[i].point - hitPoint);

					float3 R = reflect(-L, N);
					float3 shadowPoint;
					float3 shadowNormal;
					int shadowIndex;
					float shadowDepth;

					float lightDistance = magnitude(lights[i].point - hitPoint);

					cast(L, hitOffset, tris, numTris, shadowPoint, shadowNormal, shadowIndex, shadowDepth);
					bool shadow = shadowDepth != -1 && shadowDepth < lightDistance;
					if(!shadow){
						diffuse += tris[hitIndex].color * lights[i].intensity *  max(0.0f, dot(N, -L));
						specular += lights[i].color * lights[i].intensity * pow(max(0.0f,dot(R, dir)), tris[hitIndex].shinyness);
					}
				}

				float3 W = reflect(V, N);
				int reflectColor = 0;
				raytrace(W, hitOffset, tris, numTris, lights, numLights, reflectColor, step + 1);
				float r = ((reflectColor >> 16) & 0xff);
				float g = ((reflectColor >>  8) & 0xff);
				float b = ((reflectColor      ) & 0xff);
				float3 reflectedLight = tris[hitIndex].reflectivity * make_float3(r, g, b);

				float3 directLight = tris[hitIndex].color * tris[hitIndex].ambient + (diffuse * tris[hitIndex].diffuse) + (specular * tris[hitIndex].specular);

				float3 colorVec = directLight + reflectedLight;

				color = createRGB(min(int(colorVec.x), 255), min(int(colorVec.y), 255), min(int(colorVec.z), 255));
			}
		}
	}

	__device__ void cast(float3 dir, float3 origin, Triangle tris[], int numTris, float3 &hitPoint, float3 &hitNormal, int &hitIndex, float &depth){
		float closestDepth = 9999999;
		float3 point;
		float3 normal;
		depth = -1;
		
		for(int i = 0; i < numTris; i++){
			float depth_ = tris[i].intersect(dir, origin, point, normal);
			if(depth_ > 0 && depth_ < closestDepth){
				closestDepth = depth_;
				depth = depth_;
				hitPoint = point;
				hitIndex = i;
				hitNormal = normal;
			} 
		}
	}
}